#include <algorithm>
#include <vector>

#include "caffe/layers/batch_euclidean_map_layer.hpp"

namespace caffe {
  template <typename Dtype>
  void BatchEuclideanMapLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom, const vector<Blob<Dtype>*>& top){
  	const Dtype* bottom_data = bottom[0]->gpu_data();
    Dtype* top_data = top[0]->mutable_gpu_data();
    Dtype* diff_feat_data = diff_feat.mutable_gpu_data();

    max_d = Dtype(0.0);
    for (int n = 0; n < num_; ++n){
      for (int nn = 0; nn < num_; ++nn){
      	//diff_feat = x_n - x_nn
      	caffe_gpu_sub(channels_, bottom_data+n*channels_, bottom_data+nn*channels_, diff_feat_data);
      	//sim = diff_feat * diff_feat
      	Dtype distance = Dtype(0.0);
		caffe_gpu_dot(channels_, diff_feat_data, diff_feat_data, &distance);
		if (distance > max_d){
			max_d = distance;
		}
      	//top[n, nn] = sim
      	caffe_gpu_set(1, distance, top_data+n*num_+nn);
      }
    }
	caffe_gpu_scal(num_*num_, Dtype(1.0) / max_d, top_data);
  }

  template <typename Dtype>
  void BatchEuclideanMapLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top, const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom){
  	if (!propagate_down[0]){return;}
  	const Dtype* top_diff = top[0]->gpu_diff();
  	Dtype* bottom_diff = bottom[0]->mutable_gpu_diff();
  	caffe_gpu_set(num_*channels_, Dtype(0.0), bottom_diff);
  	Dtype* diff_feat_data = diff_feat.mutable_gpu_data();
  	const Dtype* bottom_data = bottom[0]->gpu_data();
  	Dtype scale = Dtype(0.0);
  	for (int n = 0; n < num_; ++n){
      for (int nn = 0; nn < num_; ++nn){
      	caffe_gpu_sub(channels_, bottom_data+n*channels_, bottom_data+nn*channels_, diff_feat_data);
        caffe_copy(1, top_diff+n*num_+nn, &scale);
		//const Dtype* diff_feat_data2 = diff_feat.gpu_data();
        caffe_gpu_axpy(channels_, scale*Dtype(2.0) / max_d, diff_feat_data, bottom_diff+n*channels_);
      }
  	}
  }
  INSTANTIATE_LAYER_GPU_FUNCS(BatchEuclideanMapLayer);
}
